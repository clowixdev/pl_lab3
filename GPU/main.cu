#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 16

__global__ void add_to_row(float* dev_matrix, int row, int diag, float k, int size) {
    int col = diag + (blockIdx.x * blockDim.x + threadIdx.x);

	if (col < size) {
		dev_matrix[row*size+col] += k * dev_matrix[diag*size+col];
	}
}

__global__ void compute_k(float* dev_matrix, int diag, int size) {
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x);
	int row = diag + 1 + (blockIdx.x * blockDim.x + threadIdx.x);

	if (row < size) {
		float k = (- 1.0) * dev_matrix[row*size+diag] / dev_matrix[diag*size+diag];
		__syncthreads();
		add_to_row<<<blocksPerGrid, threadsPerBlock>>>(dev_matrix, row, diag, k, size);
	}
}

void triangular_matrix(float* dev_matrix, int size) {
	for (int diag = 0; diag < size-1; diag++) {
		compute_k<<<1, size - (diag + 1)>>>(dev_matrix, diag, size);
	}
}

void show_matrix(float* hst_matrix, int size) {
    for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%8.2f", hst_matrix[i*size+j]);
		}
		printf("\n");
	}
	printf("\n");
}

void fill_matrix(float* dev_matrix, int size) {
    hiprandGenerator_t gen;

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)clock());
    hiprandGenerateUniform(gen, dev_matrix, (size*size));

    hiprandDestroyGenerator(gen);
}

int main(void) {	
    int size;

	printf("Enter size of square matrix, which will be transformed into a triangular >>> ");
	fflush(stdin);
	fscanf(stdin, "%d", &size);

	float *hst_matrix;
	float *dev_matrix;

	clock_t start = clock(); //! START

	printf("allocation...");
    hst_matrix = (float*)calloc(size * size, sizeof(float));
    hipMalloc((void **)&dev_matrix, sizeof(float) * size * size);
	printf(" - allocated.\n");

	printf("filling...");
    fill_matrix(dev_matrix, size);
	printf(" - filled.\n");

	if (size <= 20) {
		hipMemcpy(hst_matrix, dev_matrix, sizeof(float) * size * size, hipMemcpyDeviceToHost);
		printf("Generated matrix:\n");
		show_matrix(hst_matrix, size);
	}

	printf("transforming...");
	triangular_matrix(dev_matrix, size);
	printf(" - transformed\n");

    if (size <= 20) {
		hipMemcpy(hst_matrix, dev_matrix, sizeof(float) * size * size, hipMemcpyDeviceToHost);
		printf("Transformed matrix:\n");
        show_matrix(hst_matrix, size);
    }

	clock_t end = clock(); //! STOP

	double ms_duration = (double)(end - start) / CLOCKS_PER_SEC * 1000;
	printf("Time to execute - %f ms\n", ms_duration);

	free(hst_matrix);
    hipFree(dev_matrix);

	return 0;
}